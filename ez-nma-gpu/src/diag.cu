#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <algorithm>
#include <functional>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/inner_product.h>
#include <thrust/copy.h>
#include "../include/diag.h"
#include "../include/build.h"
#include "../include/main.h"

/* Calculate index for each processing unit using Chess Tournament (CT) algorithm. */
__device__ void calc_index_ij(integer bid, integer iter, integer LD, integer& i, integer& j) {
	integer LD_1 = LD - 1;
	integer index1 = (bid+iter) % LD_1;
	integer index2 = (0==bid) ? LD_1 : (LD_1-bid+iter) % LD_1;
	i = min(index1, index2);
	j = max(index1, index2);
}

/* Givens matrix pre-multiplies Hessian. */
__global__ void jacobi_kernel_step1 (real* H, real* M, real* S, real* C, integer LD, integer iter, real sweep_thresh, integer stride) {
	integer bid = blockIdx.x;
	real s = S[bid];
	real c = C[bid];
	integer i, j;
	calc_index_ij(bid, iter, LD, i, j);
	real H_ji = H[j*LD+i];
	for(integer tid=threadIdx.x; tid<LD; tid+=stride) {
		real H_ik = H[i*LD+tid];
		real H_jk = H[j*LD+tid];
		if(H_ji*H_ji < sweep_thresh) {
			// inefficient global memory access pattern. should be improved.
			M[tid*LD+i] = H_ik;
			M[tid*LD+j] = H_jk;
		} else {
			// Update rows [i] and [j] for Hessian matrix (pre-multiply).
			M[tid*LD+i] =  c*H_ik + s*H_jk;
			M[tid*LD+j] = -s*H_ik + c*H_jk;
		}
	}
}

/* Givens matrix post-multiplies Hessian. Also calculate eigenvectors. */
__global__ void jacobi_kernel_step2 (real* H, real* M, real* S, real* C, integer LD, integer iter, real sweep_thresh, integer stride, real* E) {
	integer bid = blockIdx.x;
	real s = S[bid];
	real c = C[bid];
	integer i, j;
	calc_index_ij(bid, iter, LD, i, j);
	real H_ji = H[j*LD+i];
	__syncthreads();
	for(integer tid=threadIdx.x; tid<LD; tid+=stride) {
		// M elements is stored column-wise to improve memory coalesce
		real M_ki = M[i*LD+tid];
		real M_kj = M[j*LD+tid];
		if(H_ji*H_ji < sweep_thresh) {
			// H is symmetric, so use H_ik(Hjk) instead of H_ki(H_kj) for memory coalesce
			H[i*LD+tid] = M_ki;
			H[j*LD+tid] = M_kj;
		} else {
			// H is symmetric, so use H_ik(H_jk) instead of H_ki(H_kj) for memory coalesce
			H[i*LD+tid] =  c*M_ki + s*M_kj;
			H[j*LD+tid] = -s*M_ki + c*M_kj;
			// Apply rotations to eigenvectors, storing in row-major.
			real E_ik = E[i*LD+tid];
			real E_jk = E[j*LD+tid];
			E[i*LD+tid] =  c*E_ik + s*E_jk;
			E[j*LD+tid] = -s*E_ik + c*E_jk;
		}
	}
}

/* Calculate S and C for next Givens rotation, quick and dirty */
__global__ void calc_param_kernel (real* H, real *S, real *C, integer LD, integer iter) {
	integer bid = blockIdx.x;
	integer i, j;
	calc_index_ij(bid, iter, LD, i, j);
	real beta = (H[j*LD+j]-H[i*LD+i]) / (2.0*H[j*LD+i]);
	real coeff = 0.5*beta / sqrt(1.0+beta*beta);
	real s = sqrt(fmax(0.5+coeff, 0.0));
	real c = sqrt(fmax(0.5-coeff, 0.0));
	S[bid] = s; 
	C[bid] = c;
}

void diag_hessian_gpu(integer LD, real tol) {
	/********************** MEMORY SPACE ALLOCATION **********************/
	const integer dev_id = 0;
	hipDeviceProp_t dev_prop;
	hipGetDeviceProperties(&dev_prop, dev_id);
	std::cout << "DiagHessian> Diagonalizing Hessian matrix on device [" << dev_prop.name 
			  << "] with computability (" << dev_prop.major << "." << dev_prop.minor << ")" 
			  << std::endl;
	std::cout << "DiagHessian> If encountering CUDA kernel launch failure, please change the computability in Makefile accordingly." << std::endl;
	// These 2 numbers will be used many times
	const integer LDSQ = LD*LD;
	const integer LDHF = LD/2;
	thrust::device_vector<real> d_H = h_H; // Hessian matrix
	thrust::device_vector<real> d_E = h_E; // Eigenvector matrix
	thrust::device_vector<real> d_M(LDSQ, 0.0); // (Auxiliary) interMediate matrix
	thrust::device_vector<real> d_S(LDHF, 0.0); // S elements array of Givens rotation matrix
	thrust::device_vector<real> d_C(LDHF, 0.0); // C elements array of Givens rotation matrix
	real* pd_H = thrust::raw_pointer_cast(d_H.data());
	real* pd_E = thrust::raw_pointer_cast(d_E.data());
	real* pd_M = thrust::raw_pointer_cast(d_M.data());
	real* pd_S = thrust::raw_pointer_cast(d_S.data());
	real* pd_C = thrust::raw_pointer_cast(d_C.data());

	/********************** KERNEL LAUNCHING CONFIGURATION **********************/
	const integer stride = dev_prop.maxThreadsPerBlock;
	// kernel launch parameters for calculating Givens rotation coefficients
	integer num_threads_param = 1;
	integer num_blocks_param  = LDHF;
	dim3 dim_block_param(num_threads_param, 1, 1);
	dim3 dim_grid_param(num_blocks_param, 1, 1);
	// kernel launch parameters for jacobi-sweep 
	integer num_threads_jacobi = stride;
	integer num_blocks_jacobi = LDHF;
	dim3 dim_block_jacobi(num_threads_jacobi, 1, 1);
	dim3 dim_grid_jacobi(num_blocks_jacobi, 1, 1);
	
	/********************** DATA INITIALIZATION **********************/
	// h_H and h_E are currently idle and can be resued as temporary data containers
	std::fill(h_E.begin(), h_E.end(), 1.0);
	for(integer i=0; i<LD; i++) h_E[i*LD+i] = 0.0;	// h_E will zero out diagonal elements of its element-wise multiplier.
	thrust::device_vector<real> d_N = h_E; // h_N as auxiliary matrix on device works like h_E.
	std::transform(h_H.cbegin(), h_H.cend(), h_E.cbegin(), h_H.begin(), std::multiplies<real>());
	real offd_sumsq = std::inner_product(h_H.cbegin(), h_H.cend(), h_H.cbegin(), 0.0);
	if (offd_sumsq < tol) return;
	std::cout << "DiagHessian> Converging index before Jacobi sweep: " << std::fixed << offd_sumsq << std::endl;
	real delta_sum = offd_sumsq;
	real sweep_thresh = 0.5*offd_sumsq/LDSQ;

	/********************** RUN JACOBI SWEEPS **********************/
	integer sweep_counter = 0;
	while(offd_sumsq > tol) {
		integer n_iter = LD-1;
		for(integer iter=0; iter<n_iter; iter++) {
			calc_param_kernel<<<dim_grid_param, dim_block_param>>>(pd_H, pd_S, pd_C, LD, iter);
			jacobi_kernel_step1<<<dim_grid_jacobi, dim_block_jacobi>>>(pd_H, pd_M, pd_S, pd_C, LD, iter, sweep_thresh, stride);
			jacobi_kernel_step2<<<dim_grid_jacobi, dim_block_jacobi>>>(pd_H, pd_M, pd_S, pd_C, LD, iter, sweep_thresh, stride, pd_E);
		}
		sweep_counter++;
		// Transform d_M into off-diagonal matrix of H and calculate the sum of squares.
		thrust::transform(d_H.cbegin(), d_H.cend(), d_N.cbegin(), d_M.begin(), thrust::multiplies<real>());
		offd_sumsq = thrust::inner_product(d_M.cbegin(), d_M.cend(), d_M.cbegin(), 0.0);
		sweep_thresh = 0.5*offd_sumsq/LDSQ;
		delta_sum -= offd_sumsq;
		std::cout << std::fixed
				  << "DiagHessian> Sweep#" << std::setw(12) << sweep_counter
				  << " | ConvergeIndex:"   << std::setw(16) << offd_sumsq
				  << " | TargetValue:"     << std::setw(12) << tol
				  << " | DeltaOffdSumsq:"  << std::setw(12) << delta_sum
				  << std::endl;
		delta_sum = offd_sumsq;
 	}
	// copy diagonal matrix and eigenvectors back to host.
	thrust::copy(d_H.begin(), d_H.end(), h_H.begin());
	thrust::copy(d_E.begin(), d_E.end(), h_E.begin());
	std::cout << "DiagHessian> Done." << std::endl << std::endl;
}
